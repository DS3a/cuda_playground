#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void kernel(int *a, int N) {
  for (int i=0; i<N; i++) {
    *a += i;
  }
}

void device_code(int *a, int N) {
  for (int i=0; i<N; i++) {
    *a += i;
  }
}

int main() {
  int *a, *d_a, N;
  N = 1000000000;
  a = (int*)malloc(sizeof(int));
  *a = 2;

  hipMalloc((void **)&d_a, sizeof(int));
  hipMemcpy(d_a, a, sizeof(int), hipMemcpyHostToDevice);
  kernel<<<1, 1>>>(d_a, N);
  //device_code(a, N);

  hipMemcpy(a, d_a, sizeof(int), hipMemcpyDeviceToHost);
  printf("the number is: %d\n", *a);

  hipFree(d_a);
  free(a);
  return 0;
}
